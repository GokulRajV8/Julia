
#include <hip/hip_runtime.h>
extern "C"
// process definition for a thread of 1 pixel
__global__ void iterate(int size, float* zReal, float* zImag, float* cReal, float* cImag, int* n) {
    // block and thread IDs to workspace coordinates
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // processing all the data within the grid bounds
    if((ix < size) && (iy < size)) {
        int i = 0;
        float rReal = 0.0f;
        float rImag = 0.0f;
        float rMag = 0.0f;
        for(; i < 500; ++i) {
            rReal = (zReal[iy * size + ix] * zReal[iy * size + ix]) - (zImag[iy * size + ix] * zImag[iy * size + ix]) + cReal[iy * size + ix];
            rImag = 2.0f * zReal[iy * size + ix] * zImag[iy * size + ix] + cImag[iy * size + ix];
            rMag = rReal * rReal + rImag * rImag;
            if(rMag >= 4)
                break;
            zReal[iy * size + ix] = rReal;
            zImag[iy * size + ix] = rImag;
        }
        n[iy * size + ix] = i;
    }
}